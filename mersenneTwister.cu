#include "hip/hip_runtime.h"
// --- Generate random numbers with cuRAND's Mersenne Twister

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>

#define BLOCKSIZE	256
#define GRIDSIZE	64

/*******************/
/* GPU ERROR CHECK */
/*******************/
#define gpuErrchk(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*******************/
/* iDivUp FUNCTION */
/*******************/
__host__ __device__ int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/*********************/
/* GENERATION KERNEL */
/*********************/
__global__ void generate_kernel(hiprandStateMtgp32 * __restrict__ state, float * __restrict__ result, const int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int k = tid; k < N; k += blockDim.x * gridDim.x)
		result[k] = hiprand_uniform(&state[blockIdx.x]);
}

/********/
/* MAIN */
/********/
int main()
{
	const int N = 217 * 123;

	// --- Allocate space for results on host
	float *hostResults = (float *)malloc(N * sizeof(float));

	// --- Allocate and initialize space for results on device 
	float *devResults; gpuErrchk(hipMalloc(&devResults, N * sizeof(float)));
	gpuErrchk(hipMemset(devResults, 0, N * sizeof(float)));

	// --- Setup the pseudorandom number generator
	hiprandStateMtgp32 *devMTGPStates; gpuErrchk(hipMalloc(&devMTGPStates, GRIDSIZE * sizeof(hiprandStateMtgp32)));
	mtgp32_kernel_params *devKernelParams; gpuErrchk(hipMalloc(&devKernelParams, sizeof(mtgp32_kernel_params)));
	CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
	//CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, GRIDSIZE, 1234));
	CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, GRIDSIZE, time(NULL)));

	// --- Generate pseudo-random sequence and copy to the host
	generate_kernel << <GRIDSIZE, BLOCKSIZE >> >(devMTGPStates, devResults, N);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(hostResults, devResults, N * sizeof(float), hipMemcpyDeviceToHost));

	// --- Print results
	//for (int i = 0; i < N; i++) {
	for (int i = 0; i < 10; i++) {
		printf("%f\n", hostResults[i]);
	}

	// --- Cleanup
	gpuErrchk(hipFree(devMTGPStates));
	gpuErrchk(hipFree(devResults));
	free(hostResults);

	return 0;
}
