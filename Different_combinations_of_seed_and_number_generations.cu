
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DSIZE 8192*16
#define nTPB 256

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line); 
        if (abort) exit(code);
    }
}

/*************************/
/* CURAND INITIALIZATION */
/*************************/
__global__ void initCurand(hiprandState *state, unsigned long seed){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void testrand1(hiprandState *state, float *a){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] = hiprand_uniform(&state[idx]);
}

__global__ void testrand2(unsigned long seed, float *a){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    a[idx] = hiprand_uniform(&state);
}

/********/
/* MAIN */
/********/
int main() {

    int n_iter = 20;

    hiprandState *devState;  gpuErrchk(hipMalloc((void**)&devState, DSIZE*sizeof(hiprandState)));

    float *d_a;             gpuErrchk(hipMalloc((void**)&d_a, DSIZE*sizeof(float)));

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i=0; i<n_iter; i++) {

        initCurand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, 1);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand1<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time for separate kernels:  %3.1f ms \n", time);

    hipEventRecord(start, 0);

    for (int i=0; i<n_iter; i++) {

        testrand2<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(1, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time for single kernels:  %3.1f ms \n", time);

    hipEventRecord(start, 0);

    for (int i=0; i<n_iter; i++) {

        initCurand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, 1);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand1<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand1<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand1<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time for separate kernels with multiple random number generation:  %3.1f ms \n", time);

    hipEventRecord(start, 0);

    for (int i=0; i<n_iter; i++) {

        testrand2<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(1, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand2<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(1, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        testrand2<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(1, d_a);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time for single kernels for multiple random number generation:  %3.1f ms \n", time);

    getchar();
}
